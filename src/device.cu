#include "device.h"
#include "host.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>

#if defined(__HIP)
#include "hip/hip_runtime.h"
#define cudax hip##x
#else
#define cudax cuda##x
#endif

#define printCudaError(cuda_returned_error_code)                               \
  { accErrorPrint((cuda_returned_error_code), __FILE__, __LINE__); }

inline void accErrorPrint(hipError_t code, const char *file, int line) {
  fprintf(stderr, "ACC Error: %s (%d) %s %d\n", hipGetErrorString(code), code,
          file, line);
}


#include "simple-kernel.cu"

static DataType *gd_vector;
static DataType *device_setup(DataType *h_vector, unsigned int v_len) {
  hipError_t device_error;
  device_error = hipMalloc(&gd_vector, sizeof(DataType) * v_len);
  if (device_error != hipSuccess) {
    printCudaError(device_error);
    return NULL;
  }

  device_error = hipMemcpy(gd_vector, h_vector, sizeof(DataType) * v_len,
                            hipMemcpyHostToDevice);
  if (device_error != hipSuccess) {
    printCudaError(device_error);
    return NULL;
  }

  return gd_vector;
}

void device_teardown(void) { hipFree(gd_vector); }


#define TPB 1024
int device_kernel_run(const struct options *opt, DataType *d_vector) {
  hipError_t device_error;

  dim3 block(TPB);
  dim3 grid((opt->number_of_threads + TPB - 1) / TPB);

#ifdef TIME_DETAILED
  double time_start;
  time_start = getCpuSeconds();
#endif
  for (unsigned int i = 0; i < opt->outer_iterations; ++i) {
    for (unsigned int k = 0; k < opt->number_of_kernels; ++k) {
      vectorIterMult<<<grid, block>>>(d_vector, opt->number_of_threads,
                                      opt->inner_iterations);
    }
  }
#ifdef TIME_DETAILED
  time_elapsed[TOTAL_LAUNCH_COST] = getCpuSeconds() - time_start;
#endif
  hipDeviceSynchronize();
#ifdef TIME_DETAILED
  time_elapsed[EXEC_TIME] = getCpuSeconds() - time_start;
#endif
  return 0;
}

static hipGraph_t g_main_graph;
static hipGraphNode_t *g_nodes;
static hipGraphExec_t g_exec_work_graph;
static hipStream_t g_stream_for_cuda_graph;

hipError_t device_graph_setup(const struct options *opt, DataType **d_vector) {
#ifdef TIME_DETAILED
  double time_start = getCpuSeconds();
#endif
  hipError_t device_error;
  device_error = hipGraphCreate(&g_main_graph, 0);
  if (hipSuccess != device_error) {
    printCudaError(device_error);
    return device_error;
  }

  dim3 block(TPB);
  dim3 grid((opt->number_of_threads + TPB - 1) / TPB);

  void *ka_kernel[] = {(void *)d_vector, (void *)&opt->number_of_threads,
                       (void *)&opt->inner_iterations};
  hipKernelNodeParams np_kernel = {0};
  np_kernel.func = (void *)vectorIterMult;
  np_kernel.gridDim = grid;
  np_kernel.blockDim = block;
  np_kernel.kernelParams = ka_kernel;

  hipGraphNode_t *last_node = NULL;
  unsigned int num_dependencies = 0;
  g_nodes = (hipGraphNode_t *)malloc(opt->number_of_kernels *
                                      sizeof(hipGraphNode_t));
  for (unsigned int i = 0; i < opt->number_of_kernels; ++i) {
    device_error = hipGraphAddKernelNode(&g_nodes[i], g_main_graph, last_node,
                                          num_dependencies, &np_kernel);

    if (hipSuccess != device_error) {
      printCudaError(device_error);
      return device_error;
    }
    last_node = &g_nodes[i];
    num_dependencies = 1;
  }

  device_error = hipGraphInstantiateWithFlags(&g_exec_work_graph, g_main_graph,
                                      0);
  if (hipSuccess != device_error) {
    printCudaError(device_error);
    return device_error;
  }

  device_error = hipStreamCreateWithFlags(&g_stream_for_cuda_graph,
                                           hipStreamNonBlocking);
  if (hipSuccess != device_error) {
    printCudaError(device_error);
    return device_error;
  }
  device_error = hipGraphUpload(g_exec_work_graph, g_stream_for_cuda_graph);
  if (hipSuccess != device_error) {
    printCudaError(device_error);
    return device_error;
  }
#ifdef TIME_DETAILED
  time_elapsed[GRAPH_CREATION] = getCpuSeconds() - time_start;
#endif
  return hipSuccess;
}

hipError_t device_graph_run(const struct options *opt) {
  hipError_t device_error;
#ifdef TIME_DETAILED
  double time_start;
  time_start = getCpuSeconds();
#endif
  for (unsigned int i = 0; i < opt->outer_iterations; ++i) {
    hipGraphLaunch(g_exec_work_graph, g_stream_for_cuda_graph);
  }
#ifdef TIME_DETAILED
  time_elapsed[TOTAL_LAUNCH_COST] = getCpuSeconds() - time_start;
#endif
  hipStreamSynchronize(g_stream_for_cuda_graph);
#ifdef TIME_DETAILED
  time_elapsed[EXEC_TIME] = getCpuSeconds() - time_start;
#endif
#ifdef MEM_CHECK
#if defined(__HIP)
  system("rocm-smi --showmeminfo vram");
#else
  system("nvidia-smi");
#endif
#endif
  return hipSuccess;
}

void device_graph_teardown(void) {
  hipStreamDestroy(g_stream_for_cuda_graph);
  hipGraphExecDestroy(g_exec_work_graph);
  hipGraphDestroy(g_main_graph);
  free(g_nodes);
}

